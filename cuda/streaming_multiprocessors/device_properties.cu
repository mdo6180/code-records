
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    /*
    * Assign values to these variables so that the output string below prints the
    * requested properties of the currently active GPU.
    */

    int deviceId;
    hipGetDevice(&deviceId);                       // `deviceId` now points to the id of the currently active GPU.

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);      // `props` now has many useful properties about the active GPU device.

    int computeCapabilityMajor = props.major;
    int computeCapabilityMinor = props.minor;
    int multiProcessorCount = props.multiProcessorCount;
    int warpSize = props.warpSize;

    /*
    * There should be no need to modify the output string below.
    */

    printf(
        "Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", 
        deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize
    );
}