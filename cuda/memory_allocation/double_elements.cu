
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 100;
  size_t size = N * sizeof(int);

  /*
   * allocation memory on both the host and device.
   */

  int *a;
  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 256;
  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  /*
   * This launch will not work until the pointer `a` is also
   * available to the device.
   */

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  /*
   * free memory in both the host and the device.
   */

  hipFree(a);
}

