
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>


inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


inline void checkCudaKernel() {
	hipError_t err;
	err = hipGetLastError(); // `cudaGetLastError` will return the error from above.
	if (err != hipSuccess)
	{
	  printf("Error: %s\n", hipGetErrorString(err));
	}
}

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  checkCuda( hipMallocManaged(&a, size) );

  init(a, N);

  // size_t threads_per_block = 2048;
  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  checkCudaKernel();

  checkCuda( hipDeviceSynchronize() );

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  checkCuda( hipFree(a) );
}

