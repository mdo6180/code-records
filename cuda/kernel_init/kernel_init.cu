
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWith(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
    const int N = 2<<24;
    size_t size = N * sizeof(float);
    
    float *a;
    float *b;
    float *c;
    
    size_t threadsPerBlock;
    size_t numberOfBlocks;
    
    /*
    * nsys should register performance changes when execution configuration
    * is updated.
    */

    int deviceId;
    hipGetDevice(&deviceId);                       // `deviceId` now points to the id of the currently active GPU.
    
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);      // `props` now has many useful properties about the active GPU device.

    int multiProcessorCount = props.multiProcessorCount;

    threadsPerBlock = 256;
    numberOfBlocks = multiProcessorCount * 32;
    printf("numberOfBlocks = %zu\n", numberOfBlocks);
    
    hipError_t addVectorsErr;
    hipError_t asyncErr;

    // Speed up initialization by launching kernels.

    initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
    
    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
    
    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));
    
    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
    
    checkElementsAre(7, c, N);
    
    hipFree(a);
    hipFree(b);
    hipFree(c);
}
